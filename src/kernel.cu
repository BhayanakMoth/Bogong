#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/RunKernel.h"
/////////////////////////////////////////////////////////////////////////////////
////! Simple kernel to modify vertex positions in sine wave pattern
////! @param data  data in global memory
/////////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float3 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float3(u, w, v);
}
__global__ void color_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float xx = x / (float)width;
	float yy = y / (float)height;
	float u = 0.5f + 0.5f*cosf(time + xx + 0);
	float v = 0.5f + 0.5f*cosf(time + yy + 2);
	float w = 0.5f + 0.5f*cosf(time + xx + 4);
	
	// write output vertex
	pos[y*width + x] = make_float4(u, v, w,1.0);
}
__global__ void grid_kernel(float3 * pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int z = blockIdx.y*blockDim.y + threadIdx.y;
	float u = pos[x + z * width].x;
	float v = pos[x + z * width].y;
	float freq = 4.0f;
	pos[x + z * width].y = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

}
void UpdateMesh(float3 *pos, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}

void UpdateColors(float4 * pos, unsigned int width, unsigned int height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	color_kernel << < grid, block >> > (pos, width, height, time);
}

void UpdateGrid(float3 * pos, unsigned int width, unsigned int height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	grid_kernel << <grid, block >> > (pos, width, height, time);
}
