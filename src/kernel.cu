#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/RunKernel.h"

__global__ void simple_vbo_kernel(float3 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float3(u, w, v);
}
__global__ void color_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float xx = x / (float)width;
	float yy = y / (float)height;
	float u = 0.5f + 0.5f*cosf(time + xx + 0);
	float v = 0.5f + 0.5f*cosf(time + yy + 2);
	float w = 0.5f + 0.5f*cosf(time + xx + 4);
	
	// write output vertex
	pos[y*width + x] = make_float4(u, v, w,1.0);
}
__global__ void grid_kernel(float3 * pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float u = pos[x + y * width].x;
	float v = pos[x + y * width].z;
	float freq = 4.0f;
	pos[x + y * width].x = u;   
	pos[x + y * width].z = v;
	pos[x + y * width].y = 0.5*(u*u-(v-time)*(v-time));

}
__global__ void gerstenerKernel(float3 * pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x /float(width);
	float v = (float)y / float(height);
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float dix = 0.707106;
	float diy = 0.707106;
	float wi = 0.7;
	float posx = u; //+ 0.5*dix*u*cos(wi*((dix*x)+(diy*v))+time);
	float posz = v;//+ 0.5*dix*v*cos(wi*((dix*u) + (diy*v)) + time);
	float posy = sin(wi*((dix*u) + (diy*v)) + time);
	pos[x + width * y] = make_float3(posx,posy,posz);

 }
void UpdateMesh(float3 *pos, unsigned int mesh_width,
	unsigned int mesh_height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}
void GerstnerTest(float3 * pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	gerstenerKernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}
void UpdateColors(float4 * pos, unsigned int width, unsigned int height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	color_kernel << < grid, block >> > (pos, width, height, time);
}

void UpdateGrid(float3 * pos, unsigned int width, unsigned int height, float time)
{
	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	grid_kernel << <grid, block >> > (pos, width, height, time);
}
