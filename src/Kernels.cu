#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../include/StableFluid/StableFluidKernels.h"

//__global__ void ColourizeKernel(float4 * colours,int width, int height,float t)
//{
//	int x = blockIdx.x*blockDim.x + threadIdx.x;
//	int y = blockIdx.y*blockDim.y + threadIdx.y;
//	float u = ;
//	colours[x + y * width].x = ;
//	colours[x + y * width].y = 0.0f;
//	colours[x + y * width].z = 0.0f;
//	colours[x + y * width].w = 1.0f;
//
//}
void WashColor(float4 * colour, unsigned int width, unsigned int height,float t)
{
	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	//ColourizeKernel << <grid, block >> > (pos, width, height, t);
}
