#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../include/StableFluid/StableFluidKernels.h"

__global__ void ColourizeKernel(float4 * surface,int width, int height)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	surface[x + y * width].x = 1.0f;
	surface[x + y * width].y = 0.0f;
	surface[x + y * width].z = 0.0f;
	surface[x + y * width].w = 1.0f;

}
void WashColor(float4 * surface, unsigned int width, unsigned int height,float t)
{
	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	ColourizeKernel << <grid, block >> > (surface, width, height);
}
