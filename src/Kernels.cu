#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../Extra/helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "../Extra/hip/hip_runtime_api.h"         // helper functions for CUDA error check
#include<iostream>
#include "../include/StableFluid/StableFluidKernels.h"

texture<float4, 2, hipReadModeElementType> texRef;
__global__ void freshSurfaceKernel(int width, int height,float4 * devPtr)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	float4 colour = tex2D(texRef,x,y);
	devPtr[x + y * width] = colour;
}
void WashColor(const hipArray * array,int width,int height)
{

	checkCudaErrors(hipBindTextureToArray(texRef,array));
	texRef.normalized = false;
	texRef.filterMode = hipFilterModeLinear;
	float4 * devVal;
	float4 * hostVal = new float4[width*height];
	checkCudaErrors(hipMalloc(&devVal, width*height * sizeof(float4)));
	std::cout << hostVal[0].z<<" ";
	hipMemcpy(hostVal,devVal,width*height*sizeof(float4),hipMemcpyDeviceToHost);
	dim3 block(16,16);
	dim3 grid(width/16,height/16);
	freshSurfaceKernel <<< grid, block >>>(width,height,devVal);
	hipUnbindTexture(texRef);
	hipFree(devVal);
	delete hostVal;
}
