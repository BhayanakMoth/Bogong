#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../Extra/helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "../Extra/hip/hip_runtime_api.h"         // helper functions for CUDA error check
#include<iostream>
#include "../include/StableFluid/StableFluidKernels.h"

surface<void,cudaSurfaceType2D> surfRef;
__global__ void TexRead(int width, int height)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	float4 colour = { 0,1.0f,0,1.0f };
	surf2Dwrite(colour, surfRef, x * 4, y);
}
void WashColor(const hipArray * array,int width,int height)
{

	checkCudaErrors(cudaBindSurfaceToArray(surfRef,array));
	dim3 block(16,16);
	dim3 grid(width/16,height/16);
	TexRead <<< grid, block >>>(width,height); 
	
}
