#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../Extra/helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "../Extra/hip/hip_runtime_api.h"         // helper functions for CUDA error check
#include<iostream>
#include "../include/StableFluid/StableFluidKernels.h"

float t = 0.0f;
surface<void, cudaSurfaceType2D> surfRef;
__global__ void TextureUpdate(int width, int height,float time)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	float4 colour = { 0.0f,1.0f,1.0f,1.0f };
	float xx =(float) x / (float)width;
	float yy = (float)y / (float)height;
	colour.x = 0.5f + 0.5f * sinf( xx + 0 + time);
	colour.y = 0.5f + 0.5f * sinf( yy + 2 + time);
	colour.z = 0.5f + 0.5f * sinf( xx + 4 + time);
	surf2Dwrite(colour, surfRef, x * sizeof(float4), y);
	
}   
void WashColor(hipArray_t array, int width, int height, float delT)
{
	t += delT;
	checkCudaErrors(cudaBindSurfaceToArray(surfRef, array));
	dim3 block(16, 16);
	dim3 grid(width/block.x, height/block.y);
	TextureUpdate <<< grid,block >>>(width,height,t); 	
	
}
