#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../Extra/helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "../Extra/hip/hip_runtime_api.h"         // helper functions for CUDA error check
#include<iostream>
#include "../include/StableFluid/StableFluidKernels.h"

texture<float4, 2, hipReadModeElementType> texRef;
__global__ void TexRead(int width, int height,float4 * devPtr)
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	float4 colour = tex2D(texRef,x,y);
	devPtr[x + y * width] = colour;
}
void WashColor(const hipArray * array,int width,int height)
{

	checkCudaErrors(hipBindTextureToArray(texRef,array));
	texRef.normalized = false;
	texRef.filterMode = hipFilterModeLinear;
	float4 * devVal;
	float4 * hostVal = new float4[width*height];
	checkCudaErrors(hipMalloc(&devVal, width*height * sizeof(float4)));	
	dim3 block(16,16);
	dim3 grid(width/16,height/16);
	TexRead <<< grid, block >>>(width,height,devVal); 
	hipMemcpy(hostVal, devVal, width*height * sizeof(float4), hipMemcpyDeviceToHost);

	std::cout << hostVal[100].x << ", "<<hostVal[100].y<<", "<<hostVal[100].z<<", "<<hostVal[100].w<<"|";
	hipUnbindTexture(texRef);
	hipFree(devVal);
	delete hostVal;
}
