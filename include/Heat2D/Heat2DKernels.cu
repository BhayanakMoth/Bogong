#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../../Extra/helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "../../Extra/hip/hip_runtime_api.h"         // helper functions for CUDA error check
#include<iostream>
#include "Heat2DKernels.hpp"
surface<void, cudaSurfaceType2D> surf;
texture<float, 2>  texConstSrc;
texture<float, 2>  texIn;
texture<float, 2>  texOut;
volatile bool dstOut = true;

__device__ unsigned char value(float n1, float n2, int hue) {
	if (hue > 360)      hue -= 360;
	else if (hue < 0)   hue += 360;

	if (hue < 60)
		return (unsigned char)(255 * (n1 + (n2 - n1)*hue / 60));
	/*if (hue < 180)
		return (unsigned char)(255 * n2);
	if (hue < 240)
		return (unsigned char)(255 * (n1 + (n2 - n1)*(240 - hue) / 60));*/
	return (unsigned char)(255 * n1);
}
__global__ void float_to_color(float4 * optr,
	const float *outSrc) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float l = outSrc[offset];
	float s = 1;
	int h = (180 + (int)(360.0f * outSrc[offset])) % 360;
	float m1, m2;

	if (l <= 0.5f)
		m2 = l * (1 + s);
	else
		m2 = l + s - l * s;
	m1 = 2 * l - m2;

	optr[offset].x = float(value(m1, m2, h + 120)) / 255.0f;
	optr[offset].y = float(value(m1, m2, h)) / 255.0f;
	optr[offset].z = float(value(m1, m2, h - 120)) / 255.0f;
	optr[offset].w = 1.0f;
}
__global__ void blend_kernel(float *dst,
	bool dstOut, float delta) {
	// map from threadIdx/BlockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float   t, l, c, r, b;
	if (dstOut) {
		t = tex2D(texIn, x, y - 1);
		l = tex2D(texIn, x - 1, y);
		c = tex2D(texIn, x, y);
		r = tex2D(texIn, x + 1, y);
		b = tex2D(texIn, x, y + 1);
	}
	else {
		t = tex2D(texOut, x, y - 1);
		l = tex2D(texOut, x - 1, y);
		c = tex2D(texOut, x, y);
		r = tex2D(texOut, x + 1, y);
		b = tex2D(texOut, x, y + 1);
	}
	dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}
__global__ void copy_const_kernel(float *iptr) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	float c = tex2D(texConstSrc, x, y);
	if (c != 0) {
		iptr[offset] = c;
	}
}
__global__ void WriteToTexture(float4 * frame_data) {
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int offset = x + y * blockDim.x * gridDim.x;
	surf2Dwrite(frame_data[offset], surf, x * sizeof(float4), y);

}
void RunKernel(hipArray_t cuArray,float4 * frame_data) {
	checkCudaErrors(cudaBindSurfaceToArray(surf, cuArray));
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	WriteToTexture << <blocks, threads >> > (frame_data);
}
struct DataBlock {
	float4 * output_frame;
	float * dev_inSrc;
	float * dev_outSrc;
	float * dev_constSrc;

	float totalTime;
	float frames;
};
DataBlock data_block;

void anim_gpu(DataBlock * data_block, float delta) {
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	for (int i = 0; i < 2; i++) {
		float *in, *out;
		if (dstOut) {
			in = data_block->dev_inSrc;
			out = data_block->dev_outSrc;
		}
		else {
			out = data_block->dev_inSrc;
			in = data_block->dev_outSrc;
		}

		copy_const_kernel << <blocks, threads >> > (in);
		blend_kernel << <blocks, threads >> > (out, dstOut, delta);
		dstOut = !dstOut;
	}
	float_to_color << <blocks, threads >> > (data_block->output_frame, data_block->dev_inSrc);

}
void init_datablock() {
	size_t size = DIM * DIM * sizeof(float);
	size_t texSize = DIM * DIM * sizeof(float4);
	checkCudaErrors(hipMalloc((void**)&data_block.output_frame, texSize));
	checkCudaErrors(hipMalloc((void**)&data_block.dev_inSrc, size));
	checkCudaErrors(hipMalloc((void**)&data_block.dev_outSrc, size));
	checkCudaErrors(hipMalloc((void**)&data_block.dev_constSrc, size));
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	checkCudaErrors(hipBindTexture2D(NULL, texConstSrc, data_block.dev_constSrc, desc, DIM, DIM, sizeof(float)*DIM));
	checkCudaErrors(hipBindTexture2D(NULL, texIn, data_block.dev_inSrc, desc, DIM, DIM, sizeof(float)*DIM));
	checkCudaErrors(hipBindTexture2D(NULL, texOut, data_block.dev_outSrc, desc, DIM, DIM, sizeof(float)*DIM));

	//FILL IT UP WITH SHIT
	float * temp = new float[DIM*DIM];

	for (int i = 0; i < DIM*DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x > 100) && (x < 150) && (y > 250) && (y < 300))
			temp[i] = MAX_TEMP+5.0f;
		else {
			temp[i] = 0.0;
		}
	}

	hipMemcpy(data_block.dev_constSrc, temp,
		size,
		hipMemcpyHostToDevice);
}
void UpdateTexture(hipArray_t data_ptr) {
	anim_gpu(&data_block, 0.0f);
	RunKernel(data_ptr, data_block.output_frame);
}