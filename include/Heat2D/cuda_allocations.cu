#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../../Extra/helper_functions.h"    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include "../../Extra/hip/hip_runtime_api.h"         // helper functions for CUDA error check
#include<iostream>
#include "cuda_allocations.hpp"

void AllocateData(void ** devPtr, void * data, size_t size) {
	hipMalloc(devPtr, size);
	if (data != nullptr) {
		checkCudaErrors(hipMemcpy(*devPtr,data,size,hipMemcpyHostToDevice));
	}
}
void FreeData(void * devPtr) {
	hipFree(devPtr);
}