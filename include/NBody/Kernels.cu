#include "hip/hip_runtime.h"
#include "NBodyKernels.h"
__device__ float3 bodybodyInteractions(float4 bi, float4 bj, float3 ai) {
	
	//3 FLOPS
	float3 r;
	r.x = bj.x - bi.x;
	r.y = bj.y - bi.y;
	r.z = bj.z - bi.z;
	//6 FLOPS
	float dist = r.x*r.x + r.y*r.y + r.z*r.z + EPS2;
	//3 FLOPS
	float distCube = dist * dist* dist;
	
	//2 FLOP
	float distInv = 1 / sqrtf(distCube);

	//1 FLOP
	float s = bj.w * distInv;

	ai.x += r.x * s;
	ai.y += r.y * s;
	ai.z += r.z * s;

	return ai;
}