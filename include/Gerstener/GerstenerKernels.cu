#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "RunKernels.h"
__global__ void color_kernel2(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	float xx = x / (float)width;
	float yy = y / (float)height;
	float u = 0.5f + 0.5f*cosf(time + xx + 0);
	float v = 0.5f + 0.5f*cosf(time + yy + 2);
	float w = 0.5f + 0.5f*cosf(time + xx + 4);

	// write output vertex
	pos[y*width + x] = make_float4(u, v, w, 1.0);
}
__global__ void gerstenerKernel2(float3 * pos, unsigned int width, unsigned int height, float amplitude, float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x / float(width);
	float v = (float)y / float(height);
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	u += 0.01f;
	v += 0.01f;
	float dix1 = -(u) / (sqrt((u*u) + (v*v)));
	float diy1 = -(v) / sqrt((u*u) + (v*v));
	float dix2 = (u - 0.45f) / (sqrt((u - 0.45f)*(u - 0.45f) + (v - 0.45f)*(v - 0.45f)));
	float diy2 = (v - 0.45f) / sqrt((u - 0.45f)*(u - 0.45f) + (v - 0.45f)*(v - 0.45f));
	float wi = 0.7;
	float q = 1.7;
	float posx = u + q * amplitude*dix1*cos((wi*((dix1)) + time) * 180 / 3.141592) + q * amplitude*dix2*cos((wi*((dix2)) + time) * 180 / 3.141592);
	float posz = v + q * amplitude*diy1*cos((wi*((diy1)) + time) * 180 / 3.141592) + q * amplitude*diy2*cos((wi*((diy2)) + time) * 180 / 3.141592);
	float posy = amplitude * sin((wi*((dix1*u) + (diy1*v)) + time) * 180 / 3.141592) + amplitude * sin((wi*((dix2*u) + (diy2*v)) + time) * 180 / 3.141592);
	pos[x + width * y] = make_float3(posx, posy, posz);

}
__global__ void multiWaveGerstenerKernelWithNormals2(float3 * pos, float3 * norms, unsigned int width, unsigned int height, WaveProp * prop, int n, float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x / float(width);
	float v = (float)y / float(height);
	u = 2.0f*u - 1.0f;
	v = 2.0f*v - 1.0f;
	u += 0.01f;
	v += 0.01f;
	float posx = u;
	float posz = v;
	float posy = 0;
	float normx = 0;
	float normy = 0;
	float normz = 0;
	for (int i = 0; i < n; i++)
	{
		float dix = 0.0f, diy = 0.0f;
		if (prop[i].isCircular == 0xFF)
		{
			float x = prop[i].x;
			float y = prop[i].y;
			x = u - x;
			y = v - y;
			dix = (x) / (sqrt(x*x + y * y));
			diy = (y) / (sqrt(y*y + y * y));
		}
		else
		{
			dix = prop[i].dirx;
			diy = prop[i].diry;
		}
		float amplitude = prop[i].amplitude;
		float q = prop[i].q;
		float wi = prop[i].w;
		float phi = prop[i].phase;
		posx += q * amplitude*dix*cos((wi*((dix)) + time * phi) * 180 / 3.141592);
		posz += q * amplitude*diy*cos((wi*((diy)) + time * phi) * 180 / 3.141592);
		posy += amplitude * sin((wi*((dix*u) + (diy*v)) + time * phi) * 180 / 3.141592);
		normx += dix * wi * amplitude * sin((wi*(dix*u + diy * v) + time * phi) * 180 / 3.141592);
		normz += diy * wi * amplitude * sin(((wi*(dix*u + diy * v)) + time * phi) * 180 / 3.141592);
		normy += q * wi * amplitude * cos((wi*(dix*u + diy * v) + time * phi) * 180 / 3.141592);
	}
	normx -= normx;
	normz -= normz;
	normy = 1 - normy;
	pos[x + width * y] = make_float3(posx, posy, posz);
	norms[x + width * y] = make_float3(normx, normy, normz);

}
__global__ void multiWaveGerstenerKernel2(float3 * pos, unsigned int width, unsigned int height, WaveProp * prop, int n, float time)
{
	unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
	float u = (float)x / float(width);
	float v = (float)y / float(height);
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	u += 0.01f;
	v += 0.01f;
	float posx = u;
	float posz = v;
	float posy = 0;
	for (int i = 0; i < n; i++)
	{
		float dix = 0.0f, diy = 0.0f;
		if (prop[i].isCircular == 0xFF)
		{
			float x = prop[i].x;
			float y = prop[i].y;
			x = u - x;
			y = v - y;
			dix = (x) / (sqrt(x*x + y * y));
			diy = (y) / (sqrt(y*y + y * y));
		}
		else
		{
			dix = prop[i].dirx;
			diy = prop[i].diry;
		}
		float amplitude = prop[i].amplitude;
		float q = prop[i].q;
		float wi = prop[i].w;
		float phi = prop[i].phase;
		posx += q * amplitude*dix*cos((wi*((dix)) + time * phi) * 180 / 3.141592);
		posz += q * amplitude*diy*cos((wi*((diy)) + time * phi) * 180 / 3.141592);
		posy += amplitude * sin((wi*((dix*u) + (diy*v)) + time * phi) * 180 / 3.141592);
	}
	pos[x + width * y] = make_float3(posx, posy, posz);
}
void GerstnerTest2(float3 * pos, unsigned int mesh_width, unsigned int mesh_height, float amplitude, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	gerstenerKernel2<< < grid, block >> > (pos, mesh_width, mesh_height, amplitude, time);
}
void GerstnerNormalTest2(float3 * pos, float3 * norms, WaveProp * prop, MeshProp mesh, int n, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh.mesh_width / block.x, mesh.mesh_height / block.y, 1);
	WaveProp * dev_ptr;
	checkCudaErrors(hipMalloc(&dev_ptr, sizeof(WaveProp)*n));
	checkCudaErrors(hipMemcpy(dev_ptr, prop, sizeof(WaveProp)*n, hipMemcpyHostToDevice));
	multiWaveGerstenerKernelWithNormals2<< <grid, block >> > (pos, norms, mesh.mesh_width, mesh.mesh_height, dev_ptr, n, time);
	getLastCudaError("Cuda Kernel Launch failed");
	checkCudaErrors(hipFree(dev_ptr));
}
void GerstnerTest2(float3 * pos, WaveProp * prop, MeshProp mesh, int n, float time)
{
	dim3 block(8, 8, 1);
	dim3 grid(mesh.mesh_width / block.x, mesh.mesh_height / block.y, 1);
	WaveProp * dev_ptr;
	checkCudaErrors(hipMalloc(&dev_ptr, sizeof(WaveProp)*n));
	checkCudaErrors(hipMemcpy(dev_ptr, prop, sizeof(WaveProp)*n, hipMemcpyHostToDevice));
	multiWaveGerstenerKernel2<< < grid, block >> > (pos, mesh.mesh_width, mesh.mesh_height, dev_ptr, n, time);
	getLastCudaError("Cuda Kernel Launch failed");
	checkCudaErrors(hipFree(dev_ptr));
}
void UpdateColors2(float4 * pos, unsigned int width, unsigned int height, float time)
{
	// execute the kernel
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	color_kernel2<< < grid, block >> > (pos, width, height, time);
}

